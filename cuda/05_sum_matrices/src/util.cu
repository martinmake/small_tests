#include <stdio.h>

#include <hip/hip_runtime.h>

#include "util.h"

void cuda_select_device(int device)
{
	hipDeviceProp_t deviceProp;
	cudaCall(hipGetDeviceProperties(&deviceProp, device));
	printf("[DEVICE:%d]: %s\n", device, deviceProp.name);
	cudaCall(hipSetDevice(device));
}
